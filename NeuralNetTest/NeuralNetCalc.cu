﻿/* -*- mode:c++; coding:utf-8-ws-dos; tab-width:4 -*- ==================== */
/* -----------------------------------------------------------------------
 * $Id: NeuralNet.cpp 2720 2018-01-02 21:21:06+09:00 nowatari $
 * ======================================================================= */

#pragma warning(disable:4819)

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string.h>

/*======================================================================
 * Affine変換
 *======================================================================*/
static __global__ void AffineForwardKernel(double *pOutput,
                                           const double *pInput,
                                           const double *pWeight,
                                           int inputNum)
{
  int o = threadIdx.x;

  for (int i = 0; i < inputNum; ++i)
    pOutput[o] += pInput[i] * pWeight[inputNum * o + i];
}

void AffineForward(double *pOutput,
                   const double *pInput,
                   const double *pWeight,
                   int inputNum,
                   int outputNum)
{
  double *pDevInput = 0;
  double *pDevWeight = 0;
  double *pDevOutput = 0;
  hipError_t cudaStatus;

  do{
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
      break;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&pDevOutput, outputNum * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMalloc failed!");
      break;
    }

    cudaStatus = hipMalloc((void**)&pDevInput, inputNum * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMalloc failed!");
      break;
    }

    cudaStatus = hipMalloc((void**)&pDevWeight, outputNum * inputNum *sizeof(double));
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMalloc failed!");
      break;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(pDevInput,
                            pInput,
                            inputNum * sizeof(double),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMemcpy failed!");
      break;
    }

    cudaStatus = hipMemcpy(pDevWeight,
                            pWeight,
                            outputNum * inputNum * sizeof(double),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMemcpy failed!");
      break;
    }

    memset(pOutput, 0, outputNum * sizeof(double));

    cudaStatus = hipMemcpy(pDevOutput,
                            pOutput,
                            outputNum * sizeof(double),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMemcpy failed!");
      break;
    }

    // Launch a kernel on the GPU with one thread for each element.
    AffineForwardKernel<<<1, outputNum>>>(pDevOutput, pDevInput, pDevWeight, inputNum);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      break;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
      break;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pOutput,
                            pDevOutput,
                            outputNum * sizeof(double),
                            hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess)
    {
      fprintf(stderr, "hipMemcpy failed!");
      break;
    }
  }while(0);

  hipFree(pDevOutput);
  hipFree(pDevInput);
  hipFree(pDevWeight);
}
